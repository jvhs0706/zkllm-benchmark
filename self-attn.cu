#include "hip/hip_runtime.h"
#include "zksoftmax.cuh"
#include "zkfc.cuh"
#include "fr-tensor.cuh"
#include "proof.cuh"
#include "commitment.cuh"
#include "rescaling.cuh"
#include <string>

FrTensor rotate_half(const FrTensor& X, uint seq_len, uint num_head, uint head_dim)
{
    auto X_T = X.transpose(seq_len * num_head, head_dim);
    auto x1 = X_T.trunc(0, X_T.size >> 1);
    auto x2 = X_T.trunc(X_T.size >> 1, X_T.size);
    return catTensors({-x2, x1}).transpose(head_dim, seq_len * num_head);
}

int main(int argc, char *argv[])
{
    string mode = argv[1];
    string input_file_name = argv[2];
    uint seq_len = std::stoi(argv[3]);
    uint num_head = std::stoi(argv[4]);
    uint head_dim = std::stoi(argv[5]);
    uint embed_dim = num_head * head_dim;
    string workdir = argv[6];
    string layer_prefix = argv[7];
    string output_file_name = argv[8];

    if (mode == "qkv_linear")
    {
        auto q_proj = create_weight(
            workdir + "/self_attn.q_proj.weight-pp.bin",
            workdir + "/" + layer_prefix + "-self_attn.q_proj.weight-int.bin",
            workdir + "/" + layer_prefix + "-self_attn.q_proj.weight-commitment.bin",
            embed_dim,
            embed_dim
        );

        auto k_proj = create_weight(
            workdir + "/self_attn.k_proj.weight-pp.bin",
            workdir + "/" + layer_prefix + "-self_attn.k_proj.weight-int.bin",
            workdir + "/" + layer_prefix + "-self_attn.k_proj.weight-commitment.bin",
            embed_dim,
            embed_dim
        );

        auto v_proj = create_weight(
            workdir + "/self_attn.v_proj.weight-pp.bin",
            workdir + "/" + layer_prefix + "-self_attn.v_proj.weight-int.bin",
            workdir + "/" + layer_prefix + "-self_attn.v_proj.weight-commitment.bin",
            embed_dim,
            embed_dim
        );
        zkFC q_layer(embed_dim, embed_dim, q_proj.weight);
        zkFC k_layer(embed_dim, embed_dim, k_proj.weight);
        zkFC v_layer(embed_dim, embed_dim, v_proj.weight);
        Rescaling q_rescale(1 << 16);
        Rescaling k_rescale(1 << 16);
        Rescaling v_rescale(1 << 16);

        FrTensor input = FrTensor::from_int_bin(input_file_name);
        auto Q = q_layer(input);
        auto Q_ = q_rescale(Q);

        auto K = k_layer(input);
        auto K_ = k_rescale(K);

        auto V = v_layer(input);
        auto V_ = v_rescale(V);
        
        q_rescale.prove(Q, Q_);
        k_rescale.prove(K, K_);
        v_rescale.prove(V, V_);

        verifyWeightClaim(k_proj, k_layer.prove(input, K)[0]);
        verifyWeightClaim(q_proj, q_layer.prove(input, Q)[0]);
        verifyWeightClaim(v_proj, v_layer.prove(input, V)[0]);

        Q_.save_int("temp_Q.bin");
        K_.save_int("temp_K.bin");
        V_.save_int("temp_V.bin");

        cout << "QKV linear proof successfully verified!" << endl;

        return 0;
    }

    else if (mode == "head")
    {
        auto Q = FrTensor::from_int_bin("temp_head_Q.bin");
        auto K = FrTensor::from_int_bin("temp_head_K.bin");
        auto X = FrTensor::matmul(Q, K.transpose(seq_len, head_dim), seq_len, head_dim, seq_len);

        zkSoftmax softmax({1<<8, 1<<20, 1<<20}, 1, 0, 1UL<<32, {1<<18, 1<<22}, seq_len, seq_len, head_dim, 1);
        Rescaling rs1(1<< 14), rs2(1<<13), rs3(1<<13);

        FrTensor shift(seq_len), X_shifted(seq_len * seq_len);
        vector<FrTensor> X_segments, Y_segments, m_segments;
        FrTensor Y = softmax.compute(X, shift, X_shifted, X_segments, Y_segments, m_segments);    
        Y.save_long("temp_head_Y.bin");
        
        auto V = FrTensor::from_int_bin("temp_head_V.bin");
        auto out = FrTensor::matmul(Y, V, seq_len, seq_len, head_dim);
        auto out_ = rs3(out);
        auto out__ = rs2(out_);
        auto out___ = rs1(out__);

        out___.save_int("temp_head_out.bin");

        rs1.prove(out__, out___);
        rs2.prove(out_, out__);
        rs3.prove(out, out_);
        auto temp_rand = random_vec(3);
        vector<Polynomial> proof;
        auto u1 = random_vec(ceilLog2(seq_len));
        auto u2 = random_vec(ceilLog2(head_dim));
        auto ud = random_vec(ceilLog2(seq_len));
        auto claim = out.multi_dim_me({u1, u2}, {seq_len, head_dim});
        auto final_claim = zkip(claim, Y.partial_me(u1, seq_len, seq_len), V.partial_me(u2, head_dim, 1), ud, proof);

        softmax.prove(Y, X, shift, X_shifted, X_segments, Y_segments, m_segments, 
        random_vec(ceilLog2(Y.size)), random_vec(ceilLog2(Y.size)), temp_rand[0], temp_rand[1], temp_rand[2], proof);
        auto u1_ = random_vec(ceilLog2(seq_len));
        auto u2_ = random_vec(ceilLog2(seq_len));
        auto ud_ = random_vec(ceilLog2(head_dim));
        auto claim_ = X.multi_dim_me({u1_, u2_}, {seq_len, seq_len});
        auto final_claim_ = zkip(claim_, Q.partial_me(u1_, seq_len, head_dim), K.partial_me(u2_, seq_len, head_dim), ud_, proof);
        cout << "Attention head proof successfully verified!" << endl; 
        return 0;
    }

    else if (mode == "o_linear")
    {
        auto o_proj = create_weight(
            workdir + "/self_attn.o_proj.weight-pp.bin",
            workdir + "/" + layer_prefix + "-self_attn.o_proj.weight-int.bin",
            workdir + "/" + layer_prefix + "-self_attn.o_proj.weight-commitment.bin",
            embed_dim,
            embed_dim
        );
        auto attn_out = FrTensor::from_int_bin("temp_attn_out.bin");
        zkFC o_layer(embed_dim, embed_dim, o_proj.weight);
        Rescaling o_rescale(1 << 16);
        
        auto O = o_layer(attn_out);
        auto O_ = o_rescale(O);
        O_.save_int(output_file_name);

        o_rescale.prove(O, O_);
        verifyWeightClaim(o_proj, o_layer.prove(attn_out, O)[0]);
        cout << "Output linear proof successfully verified!" << endl;
        return 0;


    }


    

    // auto o_proj = create_weight(
    //     workdir + "/self_attn.o_proj.weight-pp.bin",
    //     workdir + "/" + layer_prefix + "-self_attn.o_proj.weight-int.bin",
    //     workdir + "/" + layer_prefix + "-self_attn.o_proj.weight-commitment.bin",
    //     embed_dim,
    //     embed_dim
    // );

    
    // zkFC o_layer(embed_dim, embed_dim, o_proj.weight);

    // auto cos = FrTensor::from_int_bin("cos_temp.bin");
    // auto sin = FrTensor::from_int_bin("sin_temp.bin");

    // Rescaling q_rescale(1 << 16);
    // Rescaling q_new_rescale(1 << 16);
    // Rescaling k_rescale(1 << 16);
    // Rescaling k_new_rescale(1 << 16);
    // Rescaling v_rescale(1 << 16);
    // Rescaling o_rescale(1 << 16);
    // Rescaling y_rescale(1 << 16);

    // FrTensor input = FrTensor::from_int_bin(input_file_name);
    // // cout << input.size << endl;
    // auto Q = q_layer(input);
    // auto Q_ = q_rescale(Q);
    
    // auto Q_rotated = rotate_half(Q_, seq_len, num_head, head_dim);
    // auto Q_new = Q_ * cos + Q_rotated * sin;
    // auto Q_new_ = q_new_rescale(Q_new);

    // auto K = k_layer(input);
    // auto K_ = k_rescale(K);
    
    // auto K_rotated = rotate_half(K_, seq_len, num_head, head_dim);
    // auto K_new = K_ * cos + K_rotated * sin;
    // auto K_new_ = k_new_rescale(K_new);

    // auto V = v_layer(input);
    // auto V_ = v_rescale(V);
    

    // zkAttn attn(1L << 16, 1L << 16, {1 << 16, 1 << 16, 1 << 16}, 1, 0, {1.0 * (1L << 5), 1.0 * (1L << 11)}, seq_len, seq_len, embed_dim, 1 << 12);

    // // CACHES
    // FrTensor sm_in(seq_len * seq_len), sm_out(seq_len * seq_len), sm_shift(seq_len), sm_in_shifted(seq_len * seq_len);
    // vector<FrTensor> sm_in_segments, sm_out_segments, sm_m_segments;

    // auto Y = attn.compute(Q_new_, K_new_, V_, sm_in, sm_out, sm_shift, sm_in_shifted, sm_in_segments, sm_out_segments, sm_m_segments);
    
    // auto Y_ = y_rescale(Y);
    // auto O = o_layer(Y_);
    // auto O_ = o_rescale(O);
    

    // O_.save_int(output_file_name);

    // o_rescale.prove(O, O_);

    // y_rescale.prove(Y, Y_);

    // attn.prove(Q_new_, K_new_, V_, Y, sm_out, sm_in, sm_shift, sm_in_shifted, sm_in_segments, sm_out_segments, sm_m_segments);

    // q_new_rescale.prove(Q_new, Q_new_);
    // q_rescale.prove(Q, Q_);
    // hadamard_product_sumcheck(Q_, cos, random_vec(ceilLog2(Q_.size)), random_vec(ceilLog2(Q_.size)));
    // hadamard_product_sumcheck(Q_rotated, sin, random_vec(ceilLog2(Q_rotated.size)), random_vec(ceilLog2(Q_rotated.size)));
    // k_new_rescale.prove(K_new, K_new_);
    // k_rescale.prove(K, K_);
    // hadamard_product_sumcheck(K_, cos, random_vec(ceilLog2(K_.size)), random_vec(ceilLog2(K_.size)));
    // hadamard_product_sumcheck(K_rotated, sin, random_vec(ceilLog2(K_rotated.size)), random_vec(ceilLog2(K_rotated.size)));
    // v_rescale.prove(V, V_);

    // verifyWeightClaim(o_proj, o_layer.prove(Y_, O)[0]);
    // verifyWeightClaim(k_proj, k_layer.prove(input, K)[0]);
    // verifyWeightClaim(q_proj, q_layer.prove(input, Q)[0]);
    // verifyWeightClaim(v_proj, v_layer.prove(input, V)[0]);

    // cout << O_(0) << " " << O_(O_.size - 1) << endl;
    return 0;
}